#include <stdlib.h>
#include <string.h>
#include <hip/hip_runtime.h>

extern "C"
void getDeviceProps_CUDA(int id, char* DeviceName, int* integr, int* mjr, int* mnr) {
  
  // Create CUDA device properties variable
  hipDeviceProp_t prop;
  
  // Query GPU properties
  hipGetDeviceProperties(&prop, id);
  
  // Storing results
  strcpy(DeviceName, prop.name);
  *integr = prop.integrated;
  *mjr = prop.major;
  *mnr = prop.minor;
}