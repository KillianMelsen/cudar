// Basic header file
#include <stdlib.h>
#include <string.h>

// CUDA runtime
#include <hip/hip_runtime.h>

// Device properties function ==============================================================================================
extern "C"
void getDeviceProps(int* id, char* deviceName, int* integr, int* mjr, int* mnr) {
  
    // Create CUDA device properties variable
    hipDeviceProp_t prop;

    // Query GPU properties
    hipGetDeviceProperties(&prop, *id);

    // Storing results
    strcpy(deviceName, prop.name);
    *integr = prop.integrated;
    *mjr = prop.major;
    *mnr = prop.minor;
}